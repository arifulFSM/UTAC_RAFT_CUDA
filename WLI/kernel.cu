#include "hip/hip_runtime.h"
#include"kernel.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void addKernel(int* c, const int* a, const int* b) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

extern "C" void cudaAdd(int* c, const int* a, const int* b, size_t size) {
	int* dev_a = nullptr;
	int* dev_b = nullptr;
	int* dev_c = nullptr;

	// Allocate GPU buffers for three vectors (two input, one output)
	hipMalloc(&dev_a, size * sizeof(int));
	hipMalloc(&dev_b, size * sizeof(int));
	hipMalloc(&dev_c, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element
	//addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	//	copy output vector from GPU buffer to host memory
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);


}